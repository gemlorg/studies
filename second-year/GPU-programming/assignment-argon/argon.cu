#include "hip/hip_runtime.h"
#include <complex>
#include <cstdio>
#include <cstdlib>
#include <ctime>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <iostream> 

#include "argon.hpp"


__global__ void calc_forces(atoms_state *state, atom *atoms, struct block_results *help) {
    if(blockIdx.x < blockIdx.y) return;
  
  __shared__ coords guests[DIM+1];
  __shared__ forces fguests[DIM+1];
  
  forces myself{};
  atom mycords{};
  real dx, dy, dz, df_by_r2, r2, r6, r12, delj6, delj12, elj6, elj12, sig2, sig4, sig6, sig12;
  int k;
  int natoms = state->natoms;
  int rc2 = state->cutoff * state->cutoff;
  int i =  blockIdx.y*blockDim.y+threadIdx.y;
  int offset = blockIdx.x*blockDim.y;
  block_results *block = &help[blockIdx.y * ((natoms - 1) / DIM + 1) + blockIdx.x];
  
  sig2 = sigma*sigma;
  sig4 = sig2*sig2;
  sig6 = sig2*sig4;
  sig12 = sig6*sig6;

  if(offset + threadIdx.y < natoms) {
    auto import = atoms[offset + threadIdx.y];
    guests[threadIdx.y].x = import.x;
    guests[threadIdx.y].y = import.y;
    guests[threadIdx.y].z = import.z;
    fguests[threadIdx.y].fx = 0;
    fguests[threadIdx.y].fy = 0;
    fguests[threadIdx.y].fz = 0;
  }
  
  __syncthreads();
  if(i < natoms) {mycords = atoms[i];} 
  elj6 = elj12 = 0;
  
  if(state->cutoff != 0) {
    for (int j=0; j<DIM; j++) {
      k = (i+j) % DIM;
      dx = mycords.x - guests[k].x;
      dy = mycords.y - guests[k].y;
      dz = mycords.z - guests[k].z;
      r2 = dx*dx + dy*dy + dz*dz;
      if( k + offset < natoms && k + offset > i && (r2 < rc2 )) {
        r6 = r2*r2*r2;
        r12 = r6*r6;

        delj6 = sig6/r6;
        delj12 = sig12/r12;
        elj6 -= delj6;
        elj12 += delj12;
        df_by_r2 = 12 * epsilon * (delj12 - delj6) / r2;
        myself.fx += df_by_r2 * dx ;
        myself.fy += df_by_r2 * dy ;
        myself.fz += df_by_r2 * dz ;
        fguests[k].fx -= df_by_r2 * dx ;
        fguests[k].fy -= df_by_r2 * dy ;
        fguests[k].fz -= df_by_r2 * dz ;
      }
    __syncthreads();
    }
  }else {
    for (int j=0; j<DIM; j++) {
      k = (i+j) % DIM;
      dx = mycords.x - guests[k].x;
      dy = mycords.y - guests[k].y;
      dz = mycords.z - guests[k].z;
      r2 = dx*dx + dy*dy + dz*dz;
      if(k + offset < natoms && k + offset > i) {
        r6 = r2*r2*r2;
        r12 = r6 * r6;
        delj6 = sig6/r6;
        delj12 = sig12/r12;
        elj6 -= delj6;
        elj12 += delj12;
        df_by_r2 =  (delj12 - delj6) / r2;
        myself.fx += df_by_r2 * dx ;
        myself.fy += df_by_r2 * dy ;
        myself.fz += df_by_r2 * dz ;
        fguests[k].fx -= df_by_r2 * dx ;
        fguests[k].fy -= df_by_r2 * dy ;
        fguests[k].fz -= df_by_r2 * dz ;
      }
      __syncthreads();
    }
  }
  
  if(threadIdx.y + offset < natoms) {
    block->guests[threadIdx.y]= fguests[threadIdx.y];
  }
  if(i < natoms ) {
        block->hosts[threadIdx.y] = myself;
        block->elj_6[threadIdx.y] = elj6;
        block->elj_12[threadIdx.y] = elj12;
      }

}

#define MAXDIM  1000
__global__ void sum_forces(atoms_state *state, atom *atoms, struct block_results *help) {
  __shared__ real  buffer6[MAXDIM]; 
  __shared__ real  buffer12[MAXDIM]; 

  int in_block = threadIdx.y;
  int y =  blockIdx.y;
  int natoms = state->natoms;
  int step = ((natoms - 1) / DIM + 1);
  int num_atom = in_block + blockIdx.y * blockDim.y;
  real sum = 0;

  

  switch(blockIdx.x) {
    case 0:
      for(int x = 0; x < step; x++) {
        if(y <= x) {
          sum += help[y * step + x].hosts[in_block].fx;
        }
        if(y >= x) {
          sum += help[x * step + y].guests[in_block].fx;
        }

      }
      if(num_atom < natoms) {
        atoms[num_atom].fx = 12 * epsilon * sum;
      }
    break;
    case 1:
      for(int x = 0; x < step; x++) {
        if(y <= x) {
          sum += help[y * step + x].hosts[in_block].fy;
        }
        if(y >= x) {
          sum += help[x * step + y].guests[in_block].fy;
        }

      }
      if(num_atom < natoms) {
        atoms[num_atom].fy = 12 * epsilon * sum;
      }
    break;
    case 2:
      for(int x = 0; x < step; x++) {
        if(y <= x) {
          sum += help[y * step + x].hosts[in_block].fz;
        }
        if(y >= x) {
          sum += help[x * step + y].guests[in_block].fz;
        }

      }
      if(num_atom < natoms) {
        atoms[num_atom].fz = 12 * epsilon * sum;
      }
    break;
    case 3:
      for(int x = 0; x < step; x++) {
        if(y <= x) {
          sum += help[y * step + x].elj_6[in_block];
        }

      }
      if(num_atom < natoms) {
        atoms[num_atom].elj_6 = 2 * epsilon * sum;
      }
       
    break;
    case 4:
       for(int x = 0; x < step; x++) {
        if(y <= x) {
          sum += help[y * step + x].elj_12[in_block];
        }

      }
      if(num_atom < natoms) {
        atoms[num_atom].elj_12 = epsilon * sum;
      }     
  }
  



}

void simulation_state::init(int size, real dt, real cutoff, int term, int heat, int cool) {
  hipError_t status;
  this->dt = dt;
  this->size = size;
  this->natoms = 4 * size * size * size;
  this->cutoff = cutoff;
  this-> term = term;
  this->heat = heat;
  this->cool = cool;
  this->cpu_atoms = new atoms_data;
  this->gpu_atoms = new gpu_atoms_data;
  this->cpu_atoms->atoms = (atom*)calloc(this->natoms , sizeof(atom));
  this->cpu_atoms->astate = new atoms_state;
  this->cpu_atoms->astate->natoms = this->natoms;
  this->cpu_atoms->astate->cutoff = cutoff;

  status = hipMalloc ((void**)&this->gpu_atoms->atoms , this->natoms * sizeof(atom));
  if (status != hipSuccess) ERROR(hipGetErrorString(status));
  status = hipMalloc ((void**)&this->gpu_atoms->astate , sizeof(atoms_state));
  if (status != hipSuccess) ERROR(hipGetErrorString(status));
  status = hipMalloc ((void**)&this->gpu_atoms->help , pow((this->natoms - 1) / DIM + 1, 2) * sizeof(struct block_results));
  /* printf("size of result %d\n", sizeof(block_results)); */
  if (status != hipSuccess) ERROR(hipGetErrorString(status));
}

void simulation_state::build() {

  atom *curr = this->cpu_atoms->atoms;
  real mx=0, my=0, mz=0;
  real box = (0.369*sqrt(2)/2);
  
  for (int ix=0;ix<this->size;ix++) {
    for (int iy=0;iy<this->size;iy++) {
      for (int iz=0;iz<this->size;iz++) {
        curr->x = box*(0.5+ix*2);
        curr->y = box*(0.5+iy*2);
        curr->z = box*(0.5+iz*2);
        curr++;
        curr->x = box*(0.5+ix*2);
        curr->y = box*(1.5+iy*2);
        curr->z = box*(1.5+iz*2);
        curr++;
        curr->x = box*(1.5+ix*2);
        curr->y = box*(0.5+iy*2);
        curr->z = box*(1.5+iz*2);
        curr++;
        curr->x = box*(1.5+ix*2);
        curr->y = box*(1.5+iy*2);
        curr->z = box*(0.5+iz*2);
        curr++;


      }
    }
  }
  curr = this->cpu_atoms->atoms;

  for(int i = 0; i < this->natoms; i++) {
    mx += curr[i].x;
    my += curr[i].y;
    mz += curr[i].z;
  }
  mx /= this->natoms;
  my /= this->natoms;
  mz /= this->natoms;

  for(int i = 0; i < this->natoms; i++) {
    curr->x -= mx;
    curr->y -= my;
    curr->z -= mz;
    curr->fx = 0;
    curr->fy = 0;
    curr->fz = 0;
    curr->vx = 0;
    curr->vy = 0;
    curr->vz = 0;
    curr++;
  }
  /* memset(this->cpu_atoms->astate, 0, sizeof(atoms_state)); */

  DEBUG_PRINTF("build ended!:3\n");
  /* curr = this->cpu_atoms->atoms; */
  /* for(int i = 0; i < this->natoms; i++) { */
  /*   DEBUG_PRINTF("%d: [%.2f %.2f %.2f]\n", i, curr->x, curr->y, curr->z); */
  /*   curr++; */
  /* } */
     

}

void simulation_state::set_random_speeds() {
  real mvx=0, mvy=0, mvz=0, update_kinetic = 0;
  srand(0);
  atom *curr = this->cpu_atoms->atoms;
  real scale = VMAX * 2  ; 
  for(int i = 0; i < this->natoms; i++) {
    curr->vx = scale * (1.0 * rand() / RAND_MAX - 0.5);
    curr->vy = scale * (1.0 * rand() / RAND_MAX - 0.5);
    curr->vz = scale * (1.0 * rand() / RAND_MAX - 0.5);
    mvx += curr->vx;
    mvy += curr->vy;
    mvz += curr->vz;
    curr++;
  }
  mvx /= this->natoms;
  mvy /= this->natoms;
  mvz /= this->natoms;
  curr = this->cpu_atoms->atoms;
  for(int i = 0; i < this->natoms; i++) {
    curr->vx -= mvx;
    curr->vy -= mvy;
    curr->vz -= mvz;
    //DEBUG_PRINTF("v %d: [%f, %f, %f]\n", i, curr->vx, curr->vy, curr->vz);
    update_kinetic += curr->kinetic();
    curr++;
  }
  this->cpu_atoms->astate->e_kin = update_kinetic * m / 2;
  DEBUG_PRINTF("first kinetic is %f\n", this->cpu_atoms->astate->e_kin);
}
void simulation_state::update_half_velocity() {
  for(atom *curr = this->cpu_atoms->atoms; curr < this->cpu_atoms->atoms + this->natoms;curr++) {
    curr->vx += curr->fx * this->dt / m / 2;
    curr->vy += curr->fy * this->dt / m / 2;
    curr->vz += curr->fz * this->dt / m / 2;
  }
}

void simulation_state::update_positions() {
  for(atom *curr = this->cpu_atoms->atoms; curr < this->cpu_atoms->atoms + this->natoms;curr++) {
    curr->x += curr->vx * this->dt;
    curr->y += curr->vy * this->dt;
    curr->z += curr->vz * this->dt;
  }
    /* atom *curr = this->cpu_atoms->atoms; */
  /* DEBUG_PRINTF("up a 0: [%f, %f, %f]\n", curr->x, curr->y, curr->z); */

}

void simulation_state::update_kinetic() {
  real ekin_update = 0;
  for(atom *curr = this->cpu_atoms->atoms; curr < this->cpu_atoms->atoms + this->natoms;curr++) {
    ekin_update += curr->kinetic(); 
  }
  /* for(int i = 10; i < 20; i++) { */
  /*     printf("num %d is ", i); */
  /*     this->cpu_atoms->atoms[i].print(); */
  /**/
  /* } */
  this->cpu_atoms->astate->e_kin = ekin_update * m / 2;


}

void simulation_state::update_potential() {
  real elj_6_update = 0, elj_12_update = 0;
  for(atom *curr = this->cpu_atoms->atoms; curr < this->cpu_atoms->atoms + this->natoms;curr++) {
    elj_6_update += curr->elj_6;
    elj_12_update += curr->elj_12;
  }
  this->cpu_atoms->astate->elj_6 = elj_6_update;
  this->cpu_atoms->astate->elj_12 = elj_12_update;
  this->cpu_atoms->astate->update_potential();

  
}

void simulation_state::move_to_gpu() {
  hipError_t status;
  status = hipMemcpy(this->gpu_atoms->atoms, this->cpu_atoms->atoms, this->natoms *  sizeof(atom), hipMemcpyHostToDevice);
  if (status != hipSuccess) ERROR(hipGetErrorString(status));
  status = hipMemcpy( this->gpu_atoms->astate, this->cpu_atoms->astate, sizeof(atoms_state), hipMemcpyHostToDevice);
  if (status != hipSuccess) ERROR(hipGetErrorString(status));
 
}

void simulation_state::move_to_cpu() {
  hipError_t status;
  status = hipMemcpy(this->cpu_atoms->atoms, this->gpu_atoms->atoms, this->natoms *  sizeof(atom), hipMemcpyDeviceToHost);
  if (status != hipSuccess) ERROR(hipGetErrorString(status));
  status = hipMemcpy( this->cpu_atoms->astate, this->gpu_atoms->astate, sizeof(atoms_state), hipMemcpyDeviceToHost);
  if (status != hipSuccess) ERROR(hipGetErrorString(status));
}

void simulation_state::forces_cpu() {
  atom *first = this->cpu_atoms->atoms;
  real dx, dy, dz, df, r2, r6, r12, delj6, delj12, elj6, elj12, sig2, sig4, sig6, sig12;
  /* clock_t start, end; */

   
  sig2 = sigma*sigma;
  sig4 = sig2*sig2;
  sig6 = sig2*sig4;
  sig12 = sig6*sig6;


  for(int i = 0; i < this->natoms; i++) {
    first[i].fx = first[i].fy = first[i].fz = 0;
  }
  /* start = clock(); */
  for(int i = 0; i < this->natoms; i++) {
    for(int j = i + 1; j < this->natoms; j++) {
       dx = first[i].x - first[j].x;
       dy = first[i].y - first[j].y;
       dz = first[i].z - first[j].z;
       r2 = dx*dx + dy*dy + dz*dz;
       if(r2>=this->cutoff * this->cutoff && this->cutoff != 0) continue;
       r6 = r2*r2*r2;
       r12 = r6*r6;
       delj6 = sig6/r6;
       delj12 = sig12/r12;
       elj6 -= delj6;
       elj12 += delj12;
       df = 12 * epsilon * (delj12 - delj6);
       first[i].fx += df * dx / r2;
       first[j].fx -= df * dx / r2;
       first[i].fy += df * dy / r2;
       first[j].fy -= df * dy / r2;
       first[i].fz += df * dz / r2;
       first[j].fz -= df * dz / r2;
    }
  }
  /* end = clock(); */
  this->cpu_atoms->astate->elj_6 = elj6 * 2 * epsilon;
  this->cpu_atoms->astate->elj_12 = elj12 * epsilon;
  this->cpu_atoms->astate->update_potential();
  /* DEBUG_PRINTF("one step takes %.6f\n", ((double)end - start) * 1000/CLOCKS_PER_SEC); */

}
void simulation_state::baloon_force() {
  atom* first = this->cpu_atoms->atoms;
  real r;
  for(int i = 0; i < this->natoms; i++) {
    r = first[i].radius();

    if(r >= RBALOON) {
      first[i].fx -= (r - RBALOON) * first[i].x / r;
      first[i].fy -= (r - RBALOON) * first[i].y / r;
      first[i].fz -= (r - RBALOON) * first[i].z / r;
      this->cpu_atoms->astate->e_pot += (r - RBALOON) * (r - RBALOON) / 2;

    }
  }
}

void simulation_state::forces_gpu() {
    hipError_t status;
    dim3 threads(1,DIM,1);
    dim3 blocks((this->natoms-1)/DIM+1,(this->natoms-1)/DIM+1,1);
    clock_t tic, tac, toc;
    tic = clock();
    calc_forces<<<blocks, threads>>>(this->gpu_atoms->astate, this->gpu_atoms->atoms, this->gpu_atoms->help);

    status = hipDeviceSynchronize();
    if (status != hipSuccess){    ERROR( hipGetErrorString(status));}
    tac = clock();
    /* DEBUG_PRINTF("calc_forces took %.6fms\n", ((double)(tac - tic) * 1000 / CLOCKS_PER_SEC)); */

    blocks = dim3(5, (this->natoms-1)/DIM+1);
    sum_forces<<<blocks, threads>>>(this->gpu_atoms->astate, this->gpu_atoms->atoms, this->gpu_atoms->help);
      status = hipDeviceSynchronize();
    if (status != hipSuccess){    ERROR( hipGetErrorString(status));}

    toc = clock();
    /* DEBUG_PRINTF("sum_forces took %.6fms\n", ((double)(toc - tac) * 1000 / CLOCKS_PER_SEC)); */

}

void simulation_state::scale_velocity(real delta) {
  real alpha = 1 + delta / this->cpu_atoms->astate->temp() / this->heat;
  atom *first = this->cpu_atoms->atoms;
  for(int i = 0; i < this->natoms; i++) {
    first[i].vx *= alpha;
    first[i].vy *= alpha;
    first[i].vz *= alpha;
  }


}

void simulation_state::verlet_gpu() {
  this->update_half_velocity();
  this->update_positions();
  this->move_to_gpu();
  this->forces_gpu();
  this->move_to_cpu();
  this->update_potential();
  this->baloon_force();
  this->update_half_velocity();
  this->update_kinetic();

  this->cpu_atoms->astate->update_energy();
}

void simulation_state::verlet_cpu() {
  this->update_half_velocity();
  this->update_positions();
  forces_cpu();
  this->update_half_velocity();
  this->update_kinetic();
  this->cpu_atoms->astate->update_energy();
  /* auto a = this->cpu_atoms->atoms[0]; */
  /* printf("0 a,v,f : [%.4f,%.4f,%.4f,%.4f,%.4f,%.4f,%.4f,%.4f,%.4f]\n", a.x, a.y, a.z, a.vx, a.vy, a.vz, a.fx, a.fy, a.fz); */
}

void simulation_state::term_step(int step) {
  this->verlet_gpu(); 
}

void simulation_state::heat_step(int step) {
  this->verlet_gpu(); 
  this->scale_velocity(HEAT_DELTA);
}

void simulation_state::simulate() {
  FILE *CSV;
  time_t start, end;
  atoms_state first_state;
  struct atoms_state_stats term_stats; 
  this->build();
  this->set_random_speeds();
  CSV = fopen(Fname,"w");
  fprintf(CSV,"Time,Etot,Epot,Ekin,Elj12,Elj6,Temp\n");
   start = clock();
   this->cpu_atoms->astate->ex = 0;
   this->cpu_atoms->astate->exx = 0;
  for(int step = 0; step < this->term; step++) {
    this->term_step(step);
    if(step == 0)  first_state = std::move(*this->cpu_atoms->astate);
    if(step % SAVE == 0) this->cpu_atoms->astate->print_csv(CSV, step, this->dt, this->natoms);
    this->cpu_atoms->astate->update_stats();
    
  }

  end = clock();
  real term_time = ((real)(end - start )) / CLOCKS_PER_SEC;
   if(this->term > 0) term_stats = this->cpu_atoms->astate->get_stats(this->term, first_state);
  start = clock(); 
  for(int step = 0; step < this->heat; step++) {
    this->heat_step(step);
    if(step == 0)  first_state = std::move(*this->cpu_atoms->astate);
    if((step + this->term) % SAVE == 0) this->cpu_atoms->astate->print_csv(CSV, step + this->term, this->dt, this->natoms);
    this->cpu_atoms->astate->update_stats();
    
  }
  end = clock();
  printf("term took %.6f seconds\n", term_time);
  printf("term stats: e_tot_delta=%.2f e_tot_std=%.2f\n", term_stats.e_total_delta, term_stats.e_total_std);
  printf("%.3f & %.2f & %.2f  \\\\ \n", this->dt, term_stats.e_total_delta, term_stats.e_total_std);



  printf("heat took %.6f seconds\n", ((real)(end - start )) / CLOCKS_PER_SEC);


  fclose(CSV);

}


simulation_state state;


int main(int argc, char** argv){
    int  size, term, heat, cool;
    real dt, cutoff;
    
    if (argc!=7) {
      printf("Usage: %s SIZE STEP CUTOFF TERM HEAT COOL\n",argv[0]);
      exit(1);
    } else {
      size=atoi(argv[1]);
      dt=atof(argv[2]);
      cutoff=atof(argv[3]);
      term=atoi(argv[4]);
      if (argc>4) {
          heat=atoi(argv[5]);
          cool = atoi(argv[6]);
      }
    }
    state.init(size, dt, cutoff, term, heat, cool);
    state.simulate(); 
    
}

