
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#include <chrono>
#include <functional>
#include <iostream>
#define real float
using namespace std;
#define DIM 32
#define ROW 33
#define DIM32 32
#define DIM64 64
#define DIM96 96
#define DIM128 128
#define ROW33 33
#define ROW65 65
#define ROW97 97
#define ROW129 129

void Transpose(real* A, real* At, int N) {

  for (int i = 0; i < N; i++) {
    int loc = i * N;
    int loc_t;
    for (int j = 0; j < N; j++) {
      loc_t = j * N + i;
      At[loc_t] = A[loc];
      loc++;
    }
  }
}

void Multiply(real* A, real* B, real* C, int N) {
  // B is transposed
  int loc_a, loc_b, loc_c;
  //printf("In Multiply\n");
  for (int i = 0; i < N; i++) {
    for (int j = 0; j < N; j++) {
      loc_c = i * N + j;
      loc_a = i * N;
      loc_b = j * N;
      C[loc_c] = 0.0f;
      for (int k = 0; k < N; k++) {
        C[loc_c] += A[loc_a] * B[loc_b];
        loc_a++;
        loc_b++;
      }
    }
  }
}

void PrintMat(real* A, int row, int ext_row, int col, int ext_col, int N) {
  int cur_row;
  int loc;
  cur_row = row;
  for (int i = 0; i < ext_row; i++) {
    loc = cur_row * N + col;
    for (int j = 0; j < ext_col; j++) {
      printf("%f  ", A[loc + j]);
    }
    printf("\n");
    cur_row++;
  }
}

void CompareMatrices(real* A, real* B, int N) {
  int count = 0;
  real Sum = 0.0f;
  int loc = 0;

  for (int i = 0; i < N; i++) {
    for (int j = 0; j < N; j++) {
      if (A[loc] != B[loc]) {
        Sum += fabs(A[loc] - B[loc]);
        count++;
      }
      loc++;
    }
  }
  printf("Difference: %f\n", Sum);
  printf("Count: %d\n", count);
}

__global__ void MatrixMultGPUv1(real* A, real* B, real* C, int N) {
  int column = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int locA, locB;
  int k;
  // We assume that B is transposed
  real c = 0.0;
  locA = N * row;
  locB = N * column;
  if (column < N && row < N) {
    for (k = 0; k < N; k++) {
      c += A[row * N + k] * B[k * N + column];
    }
    C[row * N + column] = c;
  }
}

template <uint32_t dim>
__global__ void mult_gpu2(real* A, real* B, real* C, int N) {

// Zakładamy, ze macierz A jest transponowana
// (to oznacza, ze w pamieci jest ulozona kolumnami a nie wierszami)
// Wątki są ułozone wzdłuz kolumn macierzy A
// To oznacza, ze kazdy wątek operuje ma przypisany wiersz z macierzy A
// Kazdy blok wylicza fragment macierzy C rozmiaru dimxdim
// Lokalizacja fragmentu jest dana przez adres dwuwymiarowy bloku
//
// ZACZYNAMY KOD
// ustal adres początkowy w macierzy C

// ustal start_x jako początkową kolumnę na podstawie numeru bloku i wymiaru bloku
int start_x = blockIdx.x*dim;
// ustal start_y jako początkowy wiersz na podstawie numeru bloku i wymiaru bloku
int start_y = blockIdx.y*dim;
// ustaw my_row jako numer wiersza, którym się zajmuje wątek
int row = start_y + threadIdx.y;
//
// Teraz sprawdzamy czy nasz blok zajmuje się pełnymi kwadratowymi fragmentami, czy jest
// zlokalizowany na dole macierzy C lub na koncu wiersza.
// Dla bloków zlokalizowanych na dole częśc wątków nie będzie robiła obliczen.
// Dla bloków zlokalizowanych na koncach wierszy trzeba zrobic mniej iteracji pętli po kolumnach
// (i wczytac mniej wartosci z macierzy B do bufora)
int iter = blockIdx.x < (gridDim.x-1) ? dim : dim - (dim*gridDim.x-N);
int locA, locB, locC; // zmienne nadmiarowe - mozna je usunac upraszczając kod
int k; // licznik w zewnętrznej pętli - numer kolumny i wiersza
int n; // licznik w wewnetrznej pętli
real a; // wartośc z kolumny macierzy A, właściwa dla wątku
//__shared__ real OUT[dim*dim];
__shared__ real OUT[dim][dim]; // bufor w pamięci shared dla macierzy wynikowej
__shared__ real rowB[dim]; //  fragment wiersza z macierzy B
unsigned i = threadIdx.y; // numer wątku w bloku -> czyli lokalny
// unsigned glob_block = blockIdx.y*gridDim.x+blockIdx.x;
// unsigned glob_i = glob_block*blockDim.y+i;

//    if (glob_i==0) {
//        printf("blockIdx.x %d\n",blockIdx.x);
//        printf("gridDim.x %d\n",gridDim.x);
//        printf("blockIdx.x < gridDim.x %d\n",blockIdx.x < gridDim.x);
//        printf("iter %d\n",iter);
//    }

    // zerowanie tablicy OUT
    for (n=0;n<dim;n++) OUT[i][n]=0.0;
    __syncthreads();
    for (k=0;k<N;k++) { // zewnętrzna pętla
        // czytamy wartości z k-tej kolumny macierzy A
        // sprawdzamy czy nasz wątek ma cos do zrobienia
        //if (glob_i==0) printf("iteracja %d\n",k);
        if (row <N) {
            // jeśli tak to ustalamy skąd ma wątek ma przeczytac swoją wartośc kolumny
            // k-ta kolumna zaczyna się w lokalizacji k*N
            // wątek zajmuje się jej elementem takim jaki jest jego globalny numer  -> musi być row inaczej nie przejdzie dalej: utknie w pierwszym wierszu.
            locA = k * N + row;
            a = A[locA];
            // printf("numer wątku %d, row %d locA %d",i,row,locA);
            // printf("numer wątku %d, row %d locA %d a %lf\n",i,row,locA,a);
        }
        __syncthreads();
        // czytamy wartości z k-tego wiersza macierzy B
        // sprawdzamy czy nasz wątek ma coś do zrobienia
        if (i < iter) {
            locB = start_x + i + k*N;
            rowB[i] = B[locB];
            // k-ty wiersz zaczyna się w lokalizacji k*N
            // wątek ma wczytac z niej element o numerze start_x + lokalny numer wątku
            //printf("numer wątku %d, start_x %d locB %d rowB %lf\n",i,start_x,locB,rowB[i]);

        }
        __syncthreads();

        if (row <N) { //
            // wewnetrzna petla
            for (int n=0;n<iter;n++) {
                OUT[i][n] += a * rowB[n];
                // dodaj wartosc iloczynu kolumna z A razy wiersz z B do bufora wynikowego
                // kazdy wątek ma wartośc z kolumny w swojej zmiennej
                // wszystkie wątki operują na tym samym elemencie z wektora B
                // wszystkie wątki pracują nad tą samą kolumną w buforze
            }
        }
        __syncthreads();

    }
    __syncthreads();
    // store the final result in the global memory
    if (row < N ) { // if current thread is inside the solution
        for (int n=0; n< iter; n++ ) {
            locC = row*N + n + start_x;
            C[locC] = OUT[i][n];
            // skopiuj bufor do własciwej lokalizacji w pamieci globalnej
            // kazdy wątek kopiuje "swój" wiersz.
        }
    }
    __syncthreads();

}

template <uint32_t dim>
__global__ void mult_gpu3(real* A, real* B, real* C, int N) {
  int start_x = blockIdx.x * dim;
  int start_y = blockIdx.y * dim;

  int iter = blockIdx.x < gridDim.x ? dim : dim - (dim * gridDim.x - N);

  __shared__ real OUT[dim][dim + 1];
  __shared__ real rowB[dim];

  int i = threadIdx.y;

  for (int n = 0; n < dim; n++)
    OUT[n][i] = 0;
  __syncthreads();

  int row = start_y + i;
  for (int k = 0; k < N; ++k) {
    if (i < iter)
      rowB[i] = B[k * N + start_x + i];
    __syncthreads();

    real colA = row < N ? A[row * N + k] : 0;
    __syncthreads();

    for (int n = 0; n < dim; n++) {
      OUT[i][n] += colA * rowB[n];
    }
    __syncthreads();
  }
  __syncthreads();
  if (row < N) {
    for (int n = 0; n < dim; n++) {
      C[row * N + start_x + n] = OUT[i][n];
    }
  }
  __syncthreads();
}









template <uint32_t dim>
__global__ void mult_gpu4(real* A, real* B, real* C, int N) {
  int start_x = blockIdx.x * dim;
  int start_y = blockIdx.y * dim;

  int iter = blockIdx.x < gridDim.x ? DIM : DIM - (DIM * gridDim.x - N);

  __shared__ real OUT[dim][dim + 1];
  real rowB;

  int i = threadIdx.y;

  for (int n = 0; n < dim; n++)
    OUT[n][i] = 0;
  __syncthreads();

  int row = start_y + i;
  for (int k = 0; k < N; ++k) {
    if (i < iter)
      rowB = B[k * N + start_x + i];
    __syncthreads();

    real colA = row < N ? A[row * N + k] : 0;
    __syncthreads();

    for (int n = 0; n < dim; n++) {
      OUT[i][n] += colA * __shfl_sync(0xffffffff, rowB, n);
    }
    __syncthreads();
  }
  __syncthreads();
  if (row < N) {
    for (int n = 0; n < dim; n++) {
      C[row * N + start_x + n] = OUT[i][n];
    }
  }
  __syncthreads();
}

template <uint32_t dim>
__global__ void mult_gpu5(real* A, real* B, real* C, int N) {
  int start_x = blockIdx.x * dim;
  int start_y = blockIdx.y * dim;

  int iter = blockIdx.x < gridDim.x ? dim : dim - (dim * gridDim.x - N);

  real OUT[dim];
  real rowB;

  int i = threadIdx.y;

  for (int n = 0; n < dim; n++)
    OUT[n] = 0;
  __syncthreads();

  int row = start_y + i;
  for (int k = 0; k < N; ++k) {
    if (i < iter)
      rowB = B[k * N + start_x + i];
    __syncthreads();

    real colA = row < N ? A[row * N + k] : 0;
    __syncthreads();

    for (int n = 0; n < dim; n++) {
      OUT[n] += colA * __shfl_sync(0xffffffff, rowB, n);
    }
    __syncthreads();
  }
  __syncthreads();
  if (row < N) {
    for (int n = 0; n < dim; n++) {
      C[row * N + start_x + n] = OUT[n];
    }
  }
  __syncthreads();
}

template <int dim>
void Multiply_gpu_1(real* Agpu, real* Bgpu, real* Cgpu, int N) {
  dim3 threads(dim, dim, 1);
  dim3 blocks((N - 1) / dim + 1, (N - 1) / dim + 1, 1);

  MatrixMultGPUv1<<<blocks, threads>>>(Agpu, Bgpu, Cgpu, N);

  auto status = hipDeviceSynchronize();

  if (status != hipSuccess) {
    cout << hipGetErrorString(status) << endl;
  };
}

template <int dim>
void Multiply_gpu_2(real* Agpu, real* Bgpu, real* Cgpu, int N) {
  dim3 block_size(1, dim, 1);
  dim3 grid_size((N + dim - 1) /dim ,
                 (N + dim - 1) /dim, 1 );

  mult_gpu2<dim><<<grid_size, block_size>>>(Agpu, Bgpu, Cgpu, N);

  auto status = hipDeviceSynchronize();

  if (status != hipSuccess) {
    cout << hipGetErrorString(status) << endl;
  };
}
template <int dim>

void Multiply_gpu_3(real* Agpu, real* Bgpu, real* Cgpu, int N) {
  dim3 block_size(1, dim, 1);
  dim3 grid_size((N + dim - 1) / dim,
                 (N + dim - 1) / dim, 1);

  mult_gpu3<dim><<<grid_size, block_size>>>(Agpu, Bgpu, Cgpu, N);

  auto status = hipDeviceSynchronize();

  if (status != hipSuccess) {
    cout << hipGetErrorString(status) << endl;
  };
}
template <int dim>

void Multiply_gpu_4(real* Agpu, real* Bgpu, real* Cgpu, int N) {
  dim3 block_size(1, dim, 1);
  dim3 grid_size((N + dim - 1) / dim,
                 (N + dim - 1) / dim, 1);

  mult_gpu4<dim><<<grid_size, block_size>>>(Agpu, Bgpu, Cgpu, N);

  auto status = hipDeviceSynchronize();

  if (status != hipSuccess) {
    cout << hipGetErrorString(status) << endl;
  };
}
template <int dim>

void Multiply_gpu_5(real* Agpu, real* Bgpu, real* Cgpu, int N) {
  dim3 block_size(1, dim, 1);
  dim3 grid_size((N + dim - 1) / dim,
                 (N + dim - 1) / dim, 1);

  mult_gpu5<dim><<<grid_size, block_size>>>(Agpu, Bgpu, Cgpu, N);

  auto status = hipDeviceSynchronize();

  if (status != hipSuccess) {
    cout << hipGetErrorString(status) << endl;
  };
}

double get_avg_time_cpu(double* t, double* r, uint32_t times, real* A, real* B,
                        real* C, int N) {
  double sum = 0;
  double* result = (double*)malloc(times * sizeof(double));
  for (int i = 0; i < times; i++) {
    for (int i = 0; i < (N * N); i++) {
      A[i] = rand() * 1.0 / RAND_MAX;
      B[i] = rand() * 1.0 / RAND_MAX;
    }
    auto start = chrono::steady_clock::now();
    Multiply(A, B, C, N);
    auto stop = chrono::steady_clock::now();
    auto diff = stop - start;
    double dif = chrono::duration<double, milli>(diff).count();
    sum += dif;
    result[i] = dif;
  }
  sum /= times;
  double ssd = 0;
  for (int j = 0; j < times; j++) {
    ssd += pow((sum - result[j]), 2) / (times - 1);
  }
  ssd = pow(ssd, 0.5) / pow(times, 0.5);
  *t = sum;
  *r = ssd;
  std::cout << "time is: " << *t << "+-" << *r << std::endl;
  return sum;
}

template <int dim, class F>
double get_avg_time_gpu(uint32_t times, F mult, double cpu_t, double cpu_err,
                        real* A, real* B, real* Agpu, real* Bgpu, real* Cgpu,
                        int N) {
  double sum = 0;

  double* result = (double*)malloc(times * sizeof(double));

  for (int i = 0; i < times; i++) {

    for (int i = 0; i < (N * N); i++) {
      A[i] = rand() * 1.0 / RAND_MAX;
      B[i] = rand() * 1.0 / RAND_MAX;
    }
    //    cout << "A is: " << endl;
    //    PrintMat(A, 0, 3, 0, 3, N);
    //    cout << "B is: " << endl;
    //
    //    PrintMat(B, 0, 3, 0, 3, N);
    auto status =
        hipMemcpy(Agpu, A, N * N * sizeof(real), hipMemcpyHostToDevice);
    if (status != hipSuccess) {
      cout << hipGetErrorString(status) << endl;
    };
    status = hipMemcpy(Bgpu, B, N * N * sizeof(real), hipMemcpyHostToDevice);
    if (status != hipSuccess) {
      cout << hipGetErrorString(status) << endl;
    };
    auto start = chrono::steady_clock::now();
    mult(Agpu, Bgpu, Cgpu, N);
    auto stop = chrono::steady_clock::now();
    auto diff = stop - start;
    double dif = chrono::duration<double, milli>(diff).count();
    sum += dif;
    result[i] = dif;
    status = hipMemcpy(B, Cgpu, N * N * sizeof(real), hipMemcpyDeviceToHost);
    if (status != hipSuccess) {
      cout << hipGetErrorString(status) << endl;
    };
    //    cout << "C is: " << endl;
    //
    //    PrintMat(B, 0, 3, 0, 3, N);
  }
  sum /= times;

  double ssd = 0;
  for (int j = 0; j < times; j++) {
    ssd += pow((sum - result[j]), 2) / (times - 1);
  }
  ssd = pow(ssd, 0.5) / pow(times, 0.5);

    printf("Kernel $\\#$ &%d x 1 &%.2f+-%.2f &%.2f+-%.2f\n", dim, sum, ssd, cpu_t/sum, abs(cpu_t / sum - (cpu_t+cpu_err)/(sum-ssd)));

  return sum;
}

//printf("Matrix A:\n\n");
//PrintMat(A,0,N,0,N,N);
//printf("Matrix B\n\n");
//PrintMat(B,0,N,0,N,N);
//printf("OUT:\n\n");
//  PrintMat(C, 0, 7, 0, 7, N);

//void gpu_1(real* A, real* B, real* C, real* AT, real* BT, real* Cres,
//           real* Agpu, real* Bgpu, real* ATgpu, real* BTgpu, real* Cgpu,
//           int N) {
//  // we start business here
//
//  auto start = chrono::steady_clock::now();
//
//  auto status =
//      cudaMemcpy(Agpu, A, N * N * sizeof(real), cudaMemcpyHostToDevice);
//  if (status != cudaSuccess) {
//    cout << cudaGetErrorString(status) << endl;
//  };
//  printf("7\n");
//  status = cudaMemcpy(BTgpu, BT, N * N * sizeof(real), cudaMemcpyHostToDevice);
//  if (status != cudaSuccess) {
//    cout << cudaGetErrorString(status) << endl;
//  };
//  printf("8\n");
//  dim3 threads(DIM, DIM, 1);
//  dim3 blocks((N - 1) / DIM + 1, (N - 1) / DIM + 1, 1);
//
//  auto start2 = chrono::steady_clock::now();
//  // MatrixMultGPUv1<<<blocks, threads>>>(Agpu, BTgpu, Cgpu, N);
//
//  status = cudaDeviceSynchronize();
//  if (status != cudaSuccess) {
//    cout << cudaGetErrorString(status) << endl;
//  };
//  auto stop2 = chrono::steady_clock::now();
//  printf("10\n");
//
//  status = cudaMemcpy(Cres, Cgpu, N * N * sizeof(real), cudaMemcpyDeviceToHost);
//  if (status != cudaSuccess) {
//    cout << cudaGetErrorString(status) << endl;
//  };
//  printf("11\n");
//  auto stop = chrono::steady_clock::now();
//  auto diff = start2 - start;
//  cout << "cudaMemcpy -> GPU:  "
//       << chrono::duration<double, milli>(diff).count() << " ms" << endl;
//  diff = stop2 - start2;
//  cout << "Multiply GPU: " << chrono::duration<double, milli>(diff).count()
//       << " ms" << endl;
//  diff = stop - stop2;
//  //Multiply_gpu_1(32, 32, Agpu, BTgpu, Cgpu, N);
//  std::cout << "Averaged Multiply GPU: "
//            << get_avg_time_gpu(10, 32, 32, Multiply_gpu_1, A, BT, Agpu, BTgpu,
//                                Cgpu, N)
//            << endl;
//  cout << "cudaMemcpy -> host " << chrono::duration<double, milli>(diff).count()
//       << " ms" << endl;
//  diff = stop - start;
//  cout << "All GPU: " << chrono::duration<double, milli>(diff).count() << " ms"
//       << endl;
//
////  PrintMat(Cres, 0, 7, 0, 7, N);
//}

int main(int argc, char** argv) {

  int N;
  real *A, *AT;
  real *B, *BT;
  real *C, *Cres;
  hipError_t status;
  real* Agpu;
  real* ATgpu;
  real* Bgpu;
  real* BTgpu;
  real* Cgpu;

  if (argc != 2) {
    printf("Usage %s  N, where N is size of the square matrix\n", argv[0]);
  } else
    printf("%i\n", atoi(argv[1]));
  N = atoi(argv[1]);
  srand(777);

  A = (real*)malloc(sizeof(real) * N * N);
  AT = (real*)malloc(sizeof(real) * N * N);
  B = (real*)malloc(sizeof(real) * N * N);
  BT = (real*)malloc(sizeof(real) * N * N);
  C = (real*)malloc(sizeof(real) * N * N);
  Cres = (real*)malloc(sizeof(real) * N * N);
  status = hipMalloc((void**)&Agpu, N * N * sizeof(real));
  if (status != hipSuccess) {
    cout << hipGetErrorString(status) << endl;
  }
  status = hipMalloc((void**)&ATgpu, N * N * sizeof(real));
  if (status != hipSuccess) {
    cout << hipGetErrorString(status) << endl;
  }
  status = hipMalloc((void**)&Bgpu, N * N * sizeof(real));
  if (status != hipSuccess) {
    cout << hipGetErrorString(status) << endl;
  }
  status = hipMalloc((void**)&BTgpu, N * N * sizeof(real));
  if (status != hipSuccess) {
    cout << hipGetErrorString(status) << endl;
  }
  status = hipMalloc((void**)&Cgpu, N * N * sizeof(real));
  if (status != hipSuccess) {
    cout << hipGetErrorString(status) << endl;
  }
  int times = 10;

  //cpu(A, B, C, AT, BT, Cres, N);

  // we set a seed for the random number generator to obain reproducible OUTs

  //gpu_1(A, B, C, AT, BT, Cres, Agpu, Bgpu, ATgpu, BTgpu, Cgpu, N);

  //  std::cout << "Averaged Multiply GPU_2: "
  //            << get_avg_time_gpu(10, 32, 32, Multiply_gpu_2, A, BT, Agpu, BTgpu,
  //                                Cgpu, N)
  //            << endl;
  std::cout << " CPU: " << std::endl;
  double cpu_t;
  double cpu_err;
  get_avg_time_cpu(&cpu_t, &cpu_err, times, A, B, C, N);
  get_avg_time_cpu(&cpu_t, &cpu_err, times, A, B, C, N);
  get_avg_time_cpu(&cpu_t, &cpu_err, times, A, B, C, N);
  std::cout << " GPU1: 8X8 " << std::endl;
  get_avg_time_gpu<8>(times * 3, Multiply_gpu_1<8>, cpu_t, cpu_err, A, BT,
                      Agpu, BTgpu, Cgpu, N);
  std::cout << " GPU1: 8X8 " << std::endl;
  get_avg_time_gpu<8>(times, Multiply_gpu_1<8>, cpu_t, cpu_err, A, BT,
                      Agpu, BTgpu, Cgpu, N);
  std::cout << " GPU1: 16X16 " << std::endl;

  get_avg_time_gpu<16>(times, Multiply_gpu_1<16>, cpu_t, cpu_err, A, BT, Agpu,
                       BTgpu, Cgpu, N);

  std::cout << " GPU1: 32X32" << std::endl;
  get_avg_time_gpu<32>(times, Multiply_gpu_1<32>, cpu_t, cpu_err, A, BT, Agpu,
                       BTgpu, Cgpu, N);

  std::cout << " GPU2: 32X1 " << std::endl;
  get_avg_time_gpu<32>(times, Multiply_gpu_2<32>, cpu_t, cpu_err, A, BT, Agpu,
                       BTgpu, Cgpu, N);

  std::cout << " GPU2: 64X1" << std::endl;
  get_avg_time_gpu<64>(times, Multiply_gpu_2<64>, cpu_t, cpu_err, A, BT, Agpu,
                       BTgpu, Cgpu, N);

  std::cout << " GPU2: 96X1" << std::endl;
  get_avg_time_gpu<96>(times, Multiply_gpu_2<96>, cpu_t, cpu_err, A, BT, Agpu,
                       BTgpu, Cgpu, N);
  
  std::cout << " GPU2: 128X1" << std::endl;
  get_avg_time_gpu<128>(times, Multiply_gpu_2<96>, cpu_t, cpu_err, A, BT, Agpu,
                       BTgpu, Cgpu, N);



  std::cout << " GPU3: 32X1" << std::endl;
  get_avg_time_gpu<32>(times, Multiply_gpu_3<32>, cpu_t, cpu_err, A, BT, Agpu,
                       BTgpu, Cgpu, N);

  std::cout << " GPU3: 64X1" << std::endl;
  get_avg_time_gpu<64>(times, Multiply_gpu_3<64>, cpu_t, cpu_err, A, BT, Agpu,
                       BTgpu, Cgpu, N);

  std::cout << " GPU4: 32X1" << std::endl;

  get_avg_time_gpu<32>(times, Multiply_gpu_4<32>, cpu_t, cpu_err, A, BT, Agpu,
                       BTgpu, Cgpu, N);
 std::cout << " GPU5: 32X1" << std::endl;
  get_avg_time_gpu<32>(times, Multiply_gpu_5<32>, cpu_t, cpu_err, A, BT, Agpu,
                       BTgpu, Cgpu, N);
}
