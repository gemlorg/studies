#include "hip/hip_runtime.h"
//
//  Mandelbrot.cpp
//
//
//  Created by Witold Rudnicki
//
// Kompilacja
// c++ -o mandel_cpu Mandelbrot.cpp -L /usr/local/lib -l PNGwriter -l png
//

#define NO_FREETYPE

#include <math.h>
#include <pngwriter.h>

#include <chrono>
#include <iostream>

#include "Mandelbrot.h"
using namespace std;
#include <sys/time.h>

#include <iterator>
#include <numeric>

__global__ void computeMandelbrotDD(real X0, real Y0, real X1, real Y1, int POZ,
                                    int PION, int ITER, int* Mandel) {
  double dX = (X1 - X0) / (POZ - 1);
  double dY = (Y1 - Y0) / (PION - 1);
  double x, y, Zx, Zy, tZx;
  int SUM = 0;
  int SIZE = POZ * PION;
  //change places???
  int pion = blockIdx.x * blockDim.x + threadIdx.x;
  int poz = blockIdx.y * blockDim.y + threadIdx.y;
  int iter = 0;

  Zx = dX * poz + X0;
  Zy = dY * pion + Y0;
  x = 0;
  y = 0;
  if (pion * POZ + poz < POZ * PION) {
    while (x * x + y * y < 4 && iter < ITER) {
      double a = x * x - y * y + Zx;
      double b = x * y * 2 + Zy;
      x = a;
      y = b;
      iter++;
      SUM++;
    }
    Mandel[pion * POZ + poz] = iter;
  }
}

__global__ void computeMandelbrotD(real X0, real Y0, real X1, real Y1, int POZ,
                                   int PION, int ITER, int* Mandel) {
  double dX = (X1 - X0) / (POZ - 1);
  double dY = (Y1 - Y0) / (PION - 1);
  double x, y, Zx, Zy, tZx;
  int SUM = 0;
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int SIZE = POZ * PION;
  int pion = i % POZ;
  int poz = i / POZ;
  int iter = 0;

  Zx = dX * poz + X0;
  Zy = dY * pion + Y0;
  x = 0;
  y = 0;
  while (x * x + y * y < 4 && iter < ITER) {
    double a = x * x - y * y + Zx;
    double b = x * y * 2 + Zy;
    x = a;
    y = b;
    iter++;
    SUM++;
  }
  Mandel[pion * POZ + poz] = iter;
}

int compare(const void* a, const void* b) {
  return (*(double*)a - *(double*)b);
}

double get_time_for_creation(real X0, real Y0, real X1, real Y1, int POZ,
                             int PION, int ITER, int* Iters) {
  auto start2 = chrono::steady_clock::now();
  int SUM = computeMandelbrot(X0, Y0, X1, Y1, POZ, PION, ITER, Iters);
  auto stop = chrono::steady_clock::now();
  auto diff = stop - start2;
  double dif = chrono::duration<double, milli>(diff).count();
  return (double)dif;
}

int computeMandelbrot(real X0, real Y0, real X1, real Y1, int POZ, int PION,
                      int ITER, int* Mandel) {
  double dX = (X1 - X0) / (POZ - 1);
  double dY = (Y1 - Y0) / (PION - 1);
  double x, y, Zx, Zy, tZx;
  int SUM = 0;
  int i;
  int SIZE = POZ * PION;
  int pion, poz;
  int index = 0;
  for (pion = 0; pion < PION; pion++) {
    for (poz = 0; poz < POZ; poz++) {
      int i = 0;
      Zx = dX * poz + X0;
      Zy = dY * pion + Y0;
      x = 0;
      y = 0;
      while (x * x + y * y < 4 && i < ITER) {
        double a = x * x - y * y + Zx;
        double b = x * y * 2 + Zy;
        x = a;
        y = b;
        i++;
        SUM++;
      }
      Mandel[pion * POZ + poz] = i;
    }
  }
  return SUM;
}

void compute_cpu(real X0, real Y0, real X1, real Y1, int POZ, int PION,
                 int ITER, int out, double* a) {
  int* Iters = (int*)malloc(sizeof(int) * POZ * PION);

  //table for multiple resulrs
  double* result = (double*)malloc(20 * sizeof(double));
  double t = 0;
  double v = 0;
  double s = 0;

  double minn = 10000000;
  double maxx = 0;

  for (int i = 0; i < 20; i++) {
    result[i] =
        get_time_for_creation(X0, Y0, X1, Y1, POZ, PION, ITER, Iters) * 100;
    maxx = max(maxx, result[i]);
    minn = min(minn, result[i]);
    t += result[i];
    //     std::cout << "result #" << i << " time: " << result[i] << std::endl;
  }
  t /= 20;
  for (int i = 0; i < 20; i++) {
    v += pow((t - result[i]), 2);
  }
  v /= 19;
  s = sqrt(v) / sqrt(20);
  qsort(result, 20, sizeof(double), compare);
  if (out == 1) {
    std::cout << "average result: " << t << " +/- " << s << std::endl;
    std::cout << "min: " << minn << std::endl;
    std::cout << "mean: " << result[9] << std::endl;
  }
  a[0] = t;
  a[1] = s;
}

void compute_gpu(real X0, real Y0, real X1, real Y1, int POZ, int PION,
                 int ITER, int TIMES, int* Iters_gpu, double ORIG, double FRAC,
                 int d1, int d2) {
  double * result = (double*) malloc(TIMES * sizeof(double));
 //printf("calculating for %d D %d \n", d1, d2);
    dim3 blockSize{d1, d2, 1};
    dim3 blockCount{POZ / blockSize.x + 1, PION / blockSize.y + 1, 1};
    double sum = 0;
    for (int j = 0; j < TIMES; j++) {
      auto start2 = chrono::steady_clock::now();
      computeMandelbrotDD<<<blockCount, blockSize>>>(X0, Y0, X1, Y1, POZ, PION,
                                                     ITER, Iters_gpu);
      hipDeviceSynchronize();
      auto stop = chrono::steady_clock::now();
      auto diff = stop - start2;
      double dif = chrono::duration<double, milli>(diff).count();
      // printf("iteration #%d for %d D %d took %.2f ms\n", j,TOTAL/i, i, dif);
      result[j] = dif;
      sum += dif;
    }
    qsort(result, TIMES, sizeof(double), compare);
    sum /= TIMES;
    double ssd = 0;
    for (int j = 0; j < TIMES; j++) {
      ssd += pow((sum - result[j]), 2) / (TIMES - 1);
    }
    ssd = pow(ssd, 0.5) / pow(TIMES, 0.5);

    /* printf("result for %d D %d: \n", d1, d2); */
    /* printf("median: %.2f\n", result[TIMES / 2]); */
    /* printf("min: %.2f\n", result[0]); */
    /* printf("average: %.2f +/- %.2f\n", sum, ssd); */
   /* printf("speedup: %.2f +/- %.2f\n", ORIG / sum,  (ORIG + FRAC) / (sum - ssd) -  ORIG / sum); */
    printf("%d& %d& %.2f& %.2f& %.2f +/- %.2f& %.2f +/- %.2f \\\n", d1, d2, result[TIMES / 2], result[0], sum, ssd, ORIG/sum,  (ORIG + FRAC) / (sum - ssd) -  ORIG / sum);
 }

int main(int argc, char** argv) {
  struct timeval T0, T1;
  time_t start, end;
  struct timeval t0, t1;

  gettimeofday(&T0, NULL);
  if (argc != 9) {
    printf(
        "Wywołanie %s LD_Re, LD_Im, PG_Re, PG_Im, Poziom, Pion, Iteracje, "
        "TIMES\n ",
        argv[0]);
    exit(1);
  }
  //Ustaw obszar obliczeń {X0,Y0} - lewy dolny róg
  double X0 = atof(argv[1]);
  double Y0 = atof(argv[2]);
  //{X1,Y1} - prawy górny róg
  double X1 = atof(argv[3]);
  double Y1 = atof(argv[4]);
  //Ustal rozmiar w pikselach {POZ,PION}
  int POZ = atoi(argv[5]);
  int PION = atoi(argv[6]);
  //Ustal liczbę iteracji próbkowania {ITER}
  int ITER = atoi(argv[7]);
  int TIMES = atoi(argv[8]);

  //Zaalokuj tablicę do przechowywania wyniku

  int* Iters_gpu = nullptr;
  hipError_t status;
  status = hipMalloc((void**)&Iters_gpu, sizeof(int) * POZ * PION);
  if (status != hipSuccess) {
    std::cout << hipGetErrorString(status) << std::endl;
    return 1;
  }
for(int a = 0; a < 2; a++){
  double* result = (double*)calloc(TIMES, sizeof(double));
  double* arr = (double*)malloc(2 * sizeof(double));
  for (int k = 0; k < 12; k++) {
    compute_cpu(X0, Y0, X1, Y1, POZ / 10, PION / 10, ITER, 1, arr);
  }
  double ORIG = arr[0];
  double FRAC = arr[1];
  while(FRAC > 50) {
    compute_cpu(X0, Y0, X1, Y1, POZ / 10, PION / 10, ITER, 1, arr);
    ORIG = arr[0];
    FRAC = arr[1];
  }
  // do computations
  printf("Computations for rectangle { (%lf %lf), (%lf %lf) }\n", X0, Y0, X1,
         Y1);

  printf("1d calculations: \n");
  for (int i = 32; i <= 1024; i *= 2) {
    compute_gpu(X0, Y0, X1, Y1, POZ, PION, ITER, TIMES, Iters_gpu, ORIG, FRAC, i,
              1);


  }

  int TOTAL = 256;
  printf("2d calculations: \n");
  for (int i = 1; i <= TOTAL; i *= 2) {
    compute_gpu(X0, Y0, X1, Y1, POZ, PION, ITER, TIMES, Iters_gpu, ORIG, FRAC, TOTAL / i,
              i);

  }

  TOTAL = 1024;
  printf("2d calculations: \n");
  for (int i = 1; i <= TOTAL; i *= 2) {
  compute_gpu(X0, Y0, X1, Y1, POZ, PION, ITER, TIMES, Iters_gpu, ORIG, FRAC, TOTAL / i,
              i);
  }

  for (int k = 32; k >= 8; k /= 2) {
    compute_gpu(X0, Y0, X1, Y1, POZ, PION, ITER, TIMES, Iters_gpu, ORIG, FRAC,
                k, k);
  }

  compute_gpu(X0, Y0, X1, Y1, POZ, PION, ITER, TIMES, Iters_gpu, ORIG, FRAC, 32,
              16);

  compute_gpu(X0, Y0, X1, Y1, POZ, PION, ITER, TIMES, Iters_gpu, ORIG, FRAC, 64,
              8);

  compute_gpu(X0, Y0, X1, Y1, POZ, PION, ITER, TIMES, Iters_gpu, ORIG, FRAC, 8,
              64);

  compute_gpu(X0, Y0, X1, Y1, POZ, PION, ITER, TIMES, Iters_gpu, ORIG, FRAC, 16,
              32);

  /**/
  /*        auto start2 = chrono::steady_clock::now();  */
  /* gettimeofday(&t0,NULL); */
  /* dim3 blockSize{32, 1, 1}; */
  /* dim3 blockCount{POZ * PION / (blockSize.x * blockSize.y) + 1, 1, 1}; */
  /**/
  /* computeMandelbrot<<<blockCount, blockSize>>>(X0, Y0, X1, Y1, POZ, PION, ITER, Iters_gpu); */
  /* gettimeofday(&t1,NULL); */
  /*     end=clock(); */
  /* auto diff = stop - start2; */
  /**/
  /* cout << chrono::duration <double, milli> (diff).count() << " ms" << endl; */
  /* cout << chrono::duration <double, micro> (diff).count() << " us" << endl; */
  /* cout << chrono::duration <double, nano> (diff).count() << " ns" << endl; */
  /**/
  /* printf("\nTotal %d iterations took %lf s\n\n",1,1.0*(end-start)/CLOCKS_PER_SEC); */
  /* printf("Elapsed time %12.6lf s\n\n",(t1.tv_sec-t0.tv_sec)+1e-6*(t1.tv_usec-t0.tv_usec)); */

}
}

